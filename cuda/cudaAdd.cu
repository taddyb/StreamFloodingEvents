#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <R.h>

__global__ void add(float *a, float *b, float *c){
  *c = *b + *a;
}

extern "C" void gpuadd(float *a, float *b, float *c){
  float *da, *db, *dc;

  hipMalloc( (void**)&da, sizeof(float) );
  hipMalloc( (void**)&db, sizeof(float) );
  hipMalloc( (void**)&dc, sizeof(float) );

  hipMemcpy( da, a, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy( db, b, sizeof(float), hipMemcpyHostToDevice);

  add<<<1,1>>>(da, db, dc);

  hipMemcpy(c, dc, sizeof(float), hipMemcpyDeviceToHost);

  hipFree(da);
  hipFree(db);
  hipFree(dc);

  Rprintf("%.0f + %.0f = %.0f\n", *a, *b, *c);
}
